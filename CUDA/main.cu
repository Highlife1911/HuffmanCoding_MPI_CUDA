#include "hip/hip_runtime.h"
//Sriram Madhivanan
//Struct of Arrays
//Constant memory if dictinary goes beyond 191 bits
//Max possible shared memory
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "../Huffman/huffman.h"

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned int d_inputFileLength, unsigned int constMemoryFlag);

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned char *d_temp_overflow, unsigned int d_inputFileLength, unsigned int constMemoryFlag, 
						 unsigned int overflowPosition);

#define block_size 1024

int main(int argc, char **argv){
	unsigned int i;
	unsigned int distinctCharacterCount, combinedHuffmanNodes, inputFileLength, frequency[256];
	unsigned char *inputFileData, bitSequenceLength = 0, bitSequence[255];
	unsigned int cpu_time_used;
	FILE *inputFile;
	clock_t start, end;
	
	// start time measure
	start = clock();
	
	// read input file, get inputFileLength and data
	inputFile = fopen(argv[1], "rb");
	fseek(inputFile, 0, SEEK_END);
	inputFileLength = ftell(inputFile);
	fseek(inputFile, 0, SEEK_SET);
	inputFileData = (unsigned char *)malloc(inputFileLength * sizeof(unsigned char));
	fread(inputFileData, sizeof(unsigned char), inputFileLength, inputFile);
	fclose(inputFile);
	
	// find the frequency of each symbols
	for (i = 0; i < 256; i++){
		frequency[i] = 0;
	}
	for (i = 0; i < inputFileLength; i++){
		frequency[inputFileData[i]]++;
	}

	// initialize nodes of huffman tree
	distinctCharacterCount = 0;
	for (i = 0; i < 256; i++){
		if (frequency[i] > 0){
			huffmanTreeNode[distinctCharacterCount].count = frequency[i];
			huffmanTreeNode[distinctCharacterCount].letter = i;
			huffmanTreeNode[distinctCharacterCount].left = NULL;
			huffmanTreeNode[distinctCharacterCount].right = NULL;
			distinctCharacterCount++;
		}
	}
	
	// build tree 
	for (i = 0; i < distinctCharacterCount - 1; i++){
		combinedHuffmanNodes = 2 * i;
		sortHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
		buildHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
	}
	
	// build table having the bitSequence sequence and its length
	buildHuffmanDictionary(head_huffmanTreeNode, bitSequence, bitSequenceLength);

	printf("calling wrapper\n");
	wrapperGPU(&argv[2], inputFileData, inputFileLength);

	// calculate run duration
	end = clock();
	cpu_time_used = ((end - start)) * 1000 / CLOCKS_PER_SEC;
	printf("\nTime taken: %d seconds and %d milliseconds\n\n", cpu_time_used / 1000, cpu_time_used % 1000);

	return 0;
}