#include "hip/hip_runtime.h"
#include "header_cuda.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>

__global__ void compress(unsigned char *input, unsigned int *offset, struct table *table, unsigned char *temp, unsigned int nints)
{
	__shared__ struct table d_table[256];

	unsigned int i, j, k;
	unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(pos == 0);
		memcpy(d_table, table, 256*sizeof(struct table));
	__syncthreads();
	
	for(i = pos; i < nints; i += blockDim.x)
	{
		for(k = 0; k < d_table[input[i]].size; k++)
		{
			temp[offset[i]+k] = d_table[input[i]].bit[k];
		}
	}
	__syncthreads();
	
	for(i = pos * 8; i < offset[nints]; i += blockDim.x * 8)
	{
		for(j=0;j<8;j++)
		{
			if(temp[i+j] == 0)
			{
				input[i/8]=input[i/8] << 1;
			}
			else
			{
				input[i/8] = (input[i/8] << 1) | 1;
			}
		}
	}
	__syncthreads();
}

extern "C" void gpuCompress(unsigned int nints, unsigned char *h_input, unsigned int *h_offset, struct table *h_table)
{
	unsigned char *d_input, *d_temp;
	unsigned int *d_offset;
	struct table *d_table;
	hipError_t error;

	error = hipMalloc((void **)&d_input, nints*sizeof(unsigned char));
	if (error != hipSuccess)
			printf("erro_1: %s\n", hipGetErrorString(error));
		
	error = hipMalloc((void **)&d_offset, (nints + 1)*sizeof(unsigned int));
	if (error != hipSuccess)
			printf("erro_3: %s\n", hipGetErrorString(error));
		
		
	error = hipMalloc((void **)&d_table, 256*sizeof(table));
	if (error != hipSuccess)
			printf("erro_4: %s\n", hipGetErrorString(error));
		
		
	error = hipMalloc((void **)&d_temp, h_offset[nints]*sizeof(unsigned char));
	hipMemset(d_temp, 0, h_offset[nints]*sizeof(unsigned char));
	if (error!= hipSuccess)
			printf("erro_5: %s\n", hipGetErrorString(error));
	
	
	printf("Total GPU space: %.3fMB\n", (nints*sizeof(unsigned char) +
										//(h_offset[nints]/8)*sizeof(unsigned char) +
										(nints + 1)*sizeof(unsigned int) +
										256*sizeof(table) +
										h_offset[nints]*sizeof(unsigned char))/1000000.0);
	
	error = hipMemcpy(d_input, h_input, nints*sizeof(unsigned char), hipMemcpyHostToDevice);
	if (error!= hipSuccess)
				printf("erro_6: %s\n", hipGetErrorString(error));

			
	error = hipMemcpy(d_offset, h_offset, (nints + 1)*sizeof(unsigned int), hipMemcpyHostToDevice);
	if (error!= hipSuccess)
				printf("erro_7: %s\n", hipGetErrorString(error));
			
			
	error = hipMemcpy(d_table, h_table, 256 * sizeof(table), hipMemcpyHostToDevice);
	if (error!= hipSuccess)
				printf("erro_8: %s\n", hipGetErrorString(error));

	compress<<<1, 1024>>>(d_input, d_offset, d_table, d_temp, nints);
	
	hipMemcpy(h_input, d_input, ((h_offset[nints]/8))*sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	//DEBUG PRINT TABLE
	/*struct table *out_table = (struct table *)malloc(256*sizeof(struct table));
	hipMemcpy(out_table, d_table, 256*sizeof(table), hipMemcpyDeviceToHost);
	for(int i = 0; i < 256; i++)
		printf("%d\t%d\t\n", h_table[i].size, out_table[i].size);
	free(out_table);*/
	
	hipError_t error_final = hipGetLastError();
	if (error_final != hipSuccess)
		printf("erro_final: %s\n", hipGetErrorString(error_final));
	
	hipFree(d_input);
	hipFree(d_offset);
	hipFree(d_table);
	hipFree(d_temp);
}

