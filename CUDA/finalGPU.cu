#include "hip/hip_runtime.h"
//Sriram Madhivanan
//Struct of Arrays
//Constant memory if dictinary goes beyond 191 bits
//Max possible shared memory
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "../Huffman/huffman.h"

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned int d_inputFileLength, unsigned int constMemoryFlag);

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned char *d_temp_overflow, unsigned int d_inputFileLength, unsigned int constMemoryFlag, 
						 unsigned int overflowPosition);

#define block_size 1024
__constant__ unsigned char d_bitSequenceConstMemory[256][255];

int main(int argc, char **argv){
	unsigned int i;
	unsigned int distinctCharacterCount, combinedHuffmanNodes, inputFileLength, frequency[256];
	unsigned char *d_inputFileData, *inputFileData, *d_byteCompressedData,  bitSequenceLength = 0, bitSequence[255];
	unsigned int *d_compressedDataOffset, *compressedDataOffset, cpu_time_used;
	struct huffmanDict *d_huffmanDictionary;
	unsigned int integerOverflowFlag, integerOverflowIndex, bitPaddingFlag;
	FILE *inputFile, *compressedFile;
	hipError_t error;
	clock_t start, end;
	
	// start time measure
	start = clock();
	
	// read input file, get inputFileLength and data
	inputFile = fopen(argv[1], "rb");
	fseek(inputFile, 0, SEEK_END);
	inputFileLength = ftell(inputFile);
	fseek(inputFile, 0, SEEK_SET);
	inputFileData = (unsigned char *)malloc(inputFileLength * sizeof(unsigned char));
	fread(inputFileData, sizeof(unsigned char), inputFileLength, inputFile);
	fclose(inputFile);
	
	// find the frequency of each symbols
	for (i = 0; i < 256; i++){
		frequency[i] = 0;
	}
	for (i = 0; i < inputFileLength; i++){
		frequency[inputFileData[i]]++;
	}

	// initialize nodes of huffman tree
	distinctCharacterCount = 0;
	for (i = 0; i < 256; i++){
		if (frequency[i] > 0){
			huffmanTreeNode[distinctCharacterCount].count = frequency[i];
			huffmanTreeNode[distinctCharacterCount].letter = i;
			huffmanTreeNode[distinctCharacterCount].left = NULL;
			huffmanTreeNode[distinctCharacterCount].right = NULL;
			distinctCharacterCount++;
		}
	}
	
	// build tree 
	for (i = 0; i < distinctCharacterCount - 1; i++){
		combinedHuffmanNodes = 2 * i;
		sortHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
		buildHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
	}
	
	// build table having the bitSequence sequence and its length
	buildHuffmanDictionary(head_huffmanTreeNode, bitSequence, bitSequenceLength);

	// calculate compressed data offset - (1048576 is a safe number that will ensure there is no integer overflow in GPU, it should be minimum 8 * number of threads)
	integerOverflowFlag = 0;
	bitPaddingFlag = 0;
	compressedDataOffset = (unsigned int *)malloc((inputFileLength + 1) * sizeof(unsigned int));
	compressedDataOffset[0] = 0;
	for(i = 0; i < inputFileLength; i++){
		compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];
		if(compressedDataOffset[i + 1] + 1048576 < compressedDataOffset[i]){
			printf("Overflow error occured\n");
			integerOverflowFlag = 1;
			integerOverflowIndex = i;
			if(compressedDataOffset[i] % 8 != 0){
				bitPaddingFlag = 1;
				compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) + huffmanDictionary.bitSequenceLength[inputFileData[i]];
				compressedDataOffset[i] = compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
			}
			else{
				compressedDataOffset[i + 1] = 0;				
			}
		}
	}
	if(compressedDataOffset[inputFileLength] % 8 != 0){
		compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] + (8 - (compressedDataOffset[inputFileLength] % 8));
	}

	if(integerOverflowFlag == 0){
		long unsigned int mem_free, mem_total;
		long unsigned int mem_req;
		mem_req = 2 + (inputFileLength * sizeof(unsigned char) 
			+ (inputFileLength + 1) * sizeof(unsigned int) 
			+ sizeof(huffmanDict) 
			+ (long unsigned int)compressedDataOffset[inputFileLength] * sizeof(unsigned char))
			/(1024 * 1024);
		
		// query device memory
		error = hipMemGetInfo(&mem_free, &mem_total);
		printf("Total GPU memory: %lu\n", mem_total/(1024 * 1024));
		printf("Total GPU space available: %lu\n", mem_free/(1024 * 1024));
		printf("Total GPU space required: %lu\n", mem_req);

		if(mem_req < mem_free){
			// malloc
			error = hipMalloc((void **)&d_inputFileData, inputFileLength * sizeof(unsigned char));
			if (error != hipSuccess)
					printf("erro_1: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int));
			if (error != hipSuccess)
					printf("erro_2: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_huffmanDictionary, sizeof(huffmanDict));
			if (error != hipSuccess)
					printf("erro_3: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_byteCompressedData, (compressedDataOffset[inputFileLength]) * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_5: %s\n", hipGetErrorString(error));
	
			// memcpy
			error = hipMemcpy(d_inputFileData, inputFileData, inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_6: %s\n", hipGetErrorString(error));
			error = hipMemcpy(d_compressedDataOffset, compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_7: %s\n", hipGetErrorString(error));
			error = hipMemcpy(d_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDict), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
				
			// initialize d_byteCompressedData 
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_9: %s\n", hipGetErrorString(error));
				
			// copy constant memory
			if(constMemoryFlag == 1){
				error = hipMemcpyToSymbol(HIP_SYMBOL(d_bitSequenceConstMemory), bitSequenceConstMemory, 256 * 255 * sizeof(unsigned char));
				if (error!= hipSuccess)
					printf("erro_10: %s\n", hipGetErrorString(error));
			}
			
			// run kernel and copy output
			error = hipMemGetInfo(&mem_free, &mem_total);
			printf("Total GPU space left: %lu\n", mem_free/(1024 * 1024));
		
			compress<<<1, block_size>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, inputFileLength, constMemoryFlag);
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));

			error = hipMemcpy(inputFileData, d_inputFileData, ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
			if (error != hipSuccess)
				printf("erro_11: %s\n", hipGetErrorString(error));
			printf("%lu\n", ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char));
			
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
			
			// write src inputFileLength, header and compressed data to output file
			compressedFile = fopen(argv[2], "wb");
			fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
			fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
			fwrite(inputFileData, sizeof(unsigned char), (compressedDataOffset[inputFileLength] / 8), compressedFile);
			fclose(compressedFile);			
		}
	}
	else{
		long unsigned int mem_free, mem_total;
		long unsigned int mem_req;
		mem_req = 2 + (long unsigned int)((long unsigned int)inputFileLength * sizeof(unsigned char) 
					+ (long unsigned int)(inputFileLength + 1) * sizeof(unsigned int) 
					+ sizeof(huffmanDict) 
					+ (long unsigned int)compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char) 
					+ (long unsigned int)compressedDataOffset[inputFileLength] * sizeof(unsigned char))
					/(1024 * 1024);
		mem_req = mem_req * (1024 * 1024);

		// query device memory
		error = hipMemGetInfo(&mem_free, &mem_total);
		printf("Total GPU memory: %lu\n", mem_total/(1024 * 1024));
		printf("Total GPU space available: %lu\n", mem_free/(1024 * 1024));
		printf("Total GPU space required: %lu\n", mem_req/(1024 * 1024));

		if(mem_req < mem_free){
			unsigned char *d_byteCompressedDataOverflow;
			// malloc
			
			// allocate input file data
			error = hipMalloc((void **)&d_inputFileData, inputFileLength * sizeof(unsigned char));
			if (error != hipSuccess)
					printf("erro_1: %s\n", hipGetErrorString(error));
				
			// allocate offset 
			error = hipMalloc((void **)&d_compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int));
			if (error != hipSuccess)
					printf("erro_2: %s\n", hipGetErrorString(error));
				
			// allocate structure
			error = hipMalloc((void **)&d_huffmanDictionary, sizeof(huffmanDict));
			if (error != hipSuccess)
					printf("erro_3: %s\n", hipGetErrorString(error));
				
			// allocate bitSequence to byte storage
			error = hipMalloc((void **)&d_byteCompressedData, compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_5: %s\n", hipGetErrorString(error));
				
			error = hipMalloc((void **)&d_byteCompressedDataOverflow, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_6: %s\n", hipGetErrorString(error));
							
			// memcpy
			// copy input data
			error = hipMemcpy(d_inputFileData, inputFileData, inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_7: %s\n", hipGetErrorString(error));
				
			// copy offset
			error = hipMemcpy(d_compressedDataOffset, compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
				
			// copy structure
			error = hipMemcpy(d_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDict), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_9: %s\n", hipGetErrorString(error));
			
			// initialize d_byteCompressedData
			// initialize bitSequence to byte array to  0
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_10: %s\n", hipGetErrorString(error));	
				
			error = hipMemset(d_byteCompressedDataOverflow, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));		
				
			// copy constant memory
			if(constMemoryFlag == 1){
				error = hipMemcpyToSymbol(HIP_SYMBOL(d_bitSequenceConstMemory), bitSequenceConstMemory, 256 * 255 * sizeof(unsigned char));
				if (error!= hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));
			}
		
			// run kernel and copy output
			error = hipMemGetInfo(&mem_free, &mem_total);
			printf("Total GPU space left: %lu\n", mem_free/(1024 * 1024));
			
			// launch kernel
			compress<<<1, block_size>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, d_byteCompressedDataOverflow, inputFileLength, constMemoryFlag, integerOverflowIndex);
			
			// check status
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));
			
			// get output data
			if(bitPaddingFlag == 0){
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], &d_inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));
			}
			else{
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				unsigned char temp_huffmanTreeNode = inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1];
				
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1], &d_inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));				
				inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1] = temp_huffmanTreeNode | inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1];
			}

			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
			hipFree(d_byteCompressedDataOverflow);
			
			// write src inputFileLength, header and compressed data to output file
			compressedFile = fopen(argv[2], "wb");
			fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
			fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
			fwrite(inputFileData, sizeof(unsigned char), (compressedDataOffset[inputFileLength] / 8 + compressedDataOffset[integerOverflowIndex] / 8) - 1, compressedFile);
			fclose(compressedFile);			
		}
	}
	// calculate run duration
	end = clock();
	cpu_time_used = ((end - start)) * 1000 / CLOCKS_PER_SEC;
	printf("\nTime taken: %d seconds and %d milliseconds\n\n", cpu_time_used / 1000, cpu_time_used % 1000);

	return 0;
}