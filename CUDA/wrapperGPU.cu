#include "hip/hip_runtime.h"
//Sriram Madhivanan
//Struct of Arrays
//Constant memory if dictinary goes beyond 191 bits
//Max possible shared memory
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "../Huffman/huffman.h"

#define block_size 1024
unsigned char __constant__ d_bitSequenceConstMemory[256][255];

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned int d_inputFileLength, unsigned int constMemoryFlag);

__global__ void compress(unsigned char *d_inputFileData, unsigned int *d_compressedDataOffset, struct huffmanDict *d_huffmanDictionary, 
						 unsigned char *d_byteCompressedData, unsigned char *d_temp_overflow, unsigned int d_inputFileLength, unsigned int constMemoryFlag, 
						 unsigned int overflowPosition);

extern "C" int wrapperGPU(char **file, unsigned char *inputFileData, int inputFileLength){
	unsigned int i;
	unsigned int frequency[256];
	unsigned char *d_inputFileData, *d_byteCompressedData;
	unsigned int *d_compressedDataOffset, *compressedDataOffset;
	struct huffmanDict *d_huffmanDictionary;
	unsigned int integerOverflowFlag, integerOverflowIndex, bitPaddingFlag;
	FILE *compressedFile;
	hipError_t error;

	printf("%d\n", inputFileLength);
	// calculate compressed data offset - (1048576 is a safe number that will ensure there is no integer overflow in GPU, it should be minimum 8 * number of threads)
	integerOverflowFlag = 0;
	bitPaddingFlag = 0;
	compressedDataOffset = (unsigned int *)malloc((inputFileLength + 1) * sizeof(unsigned int));
	compressedDataOffset[0] = 0;
	for(i = 0; i < inputFileLength; i++){
		compressedDataOffset[i + 1] = huffmanDictionary.bitSequenceLength[inputFileData[i]] + compressedDataOffset[i];
		if(compressedDataOffset[i + 1] + 1048576 < compressedDataOffset[i]){
			printf("Overflow error occured\n");
			integerOverflowFlag = 1;
			integerOverflowIndex = i;
			if(compressedDataOffset[i] % 8 != 0){
				bitPaddingFlag = 1;
				compressedDataOffset[i + 1] = (compressedDataOffset[i] % 8) + huffmanDictionary.bitSequenceLength[inputFileData[i]];
				compressedDataOffset[i] = compressedDataOffset[i] + (8 - (compressedDataOffset[i] % 8));
			}
			else{
				compressedDataOffset[i + 1] = 0;				
			}
		}
	}
	if(compressedDataOffset[inputFileLength] % 8 != 0){
		compressedDataOffset[inputFileLength] = compressedDataOffset[inputFileLength] + (8 - (compressedDataOffset[inputFileLength] % 8));
	}

	printf("doing stuff 2\n");

	if(integerOverflowFlag == 0){
		long unsigned int mem_free, mem_total;
		long unsigned int mem_req;
		mem_req = 2 + (inputFileLength * sizeof(unsigned char) 
			+ (inputFileLength + 1) * sizeof(unsigned int) 
			+ sizeof(huffmanDict) 
			+ (long unsigned int)compressedDataOffset[inputFileLength] * sizeof(unsigned char))
			/(1024 * 1024);
		
		// query device memory
		error = hipMemGetInfo(&mem_free, &mem_total);
		printf("Total GPU memory: %lu\n", mem_total/(1024 * 1024));
		printf("Total GPU space available: %lu\n", mem_free/(1024 * 1024));
		printf("Total GPU space required: %lu\n", mem_req);

		if(mem_req < mem_free){
			// malloc
			error = hipMalloc((void **)&d_inputFileData, inputFileLength * sizeof(unsigned char));
			if (error != hipSuccess)
					printf("erro_1: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int));
			if (error != hipSuccess)
					printf("erro_2: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_huffmanDictionary, sizeof(huffmanDict));
			if (error != hipSuccess)
					printf("erro_3: %s\n", hipGetErrorString(error));
			error = hipMalloc((void **)&d_byteCompressedData, (compressedDataOffset[inputFileLength]) * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_5: %s\n", hipGetErrorString(error));
	
			// memcpy
			error = hipMemcpy(d_inputFileData, inputFileData, inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_6: %s\n", hipGetErrorString(error));
			error = hipMemcpy(d_compressedDataOffset, compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_7: %s\n", hipGetErrorString(error));
			error = hipMemcpy(d_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDict), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
				
			// initialize d_byteCompressedData 
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_9: %s\n", hipGetErrorString(error));
				
			// copy constant memory
			if(constMemoryFlag == 1){
				error = hipMemcpyToSymbol(HIP_SYMBOL(d_bitSequenceConstMemory), bitSequenceConstMemory, 256 * 255 * sizeof(unsigned char));
				if (error!= hipSuccess)
					printf("erro_10: %s\n", hipGetErrorString(error));
			}
			
			// run kernel and copy output
			error = hipMemGetInfo(&mem_free, &mem_total);
			printf("Total GPU space left: %lu\n", mem_free/(1024 * 1024));
		
			compress<<<1, block_size>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, inputFileLength, constMemoryFlag);
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));

			error = hipMemcpy(inputFileData, d_inputFileData, ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
			if (error != hipSuccess)
				printf("erro_11: %s\n", hipGetErrorString(error));
			printf("%lu\n", ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char));
			
			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
			
			// write src inputFileLength, header and compressed data to output file
			compressedFile = fopen(*file, "wb");
			fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
			fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
			fwrite(inputFileData, sizeof(unsigned char), (compressedDataOffset[inputFileLength] / 8), compressedFile);
			fclose(compressedFile);			
		}
	}
	else{
		long unsigned int mem_free, mem_total;
		long unsigned int mem_req;
		mem_req = 2 + (long unsigned int)((long unsigned int)inputFileLength * sizeof(unsigned char) 
					+ (long unsigned int)(inputFileLength + 1) * sizeof(unsigned int) 
					+ sizeof(huffmanDict) 
					+ (long unsigned int)compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char) 
					+ (long unsigned int)compressedDataOffset[inputFileLength] * sizeof(unsigned char))
					/(1024 * 1024);
		mem_req = mem_req * (1024 * 1024);

		// query device memory
		error = hipMemGetInfo(&mem_free, &mem_total);
		printf("Total GPU memory: %lu\n", mem_total/(1024 * 1024));
		printf("Total GPU space available: %lu\n", mem_free/(1024 * 1024));
		printf("Total GPU space required: %lu\n", mem_req/(1024 * 1024));

		if(mem_req < mem_free){
			unsigned char *d_byteCompressedDataOverflow;
			// malloc
			
			// allocate input file data
			error = hipMalloc((void **)&d_inputFileData, inputFileLength * sizeof(unsigned char));
			if (error != hipSuccess)
					printf("erro_1: %s\n", hipGetErrorString(error));
				
			// allocate offset 
			error = hipMalloc((void **)&d_compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int));
			if (error != hipSuccess)
					printf("erro_2: %s\n", hipGetErrorString(error));
				
			// allocate structure
			error = hipMalloc((void **)&d_huffmanDictionary, sizeof(huffmanDict));
			if (error != hipSuccess)
					printf("erro_3: %s\n", hipGetErrorString(error));
				
			// allocate bitSequence to byte storage
			error = hipMalloc((void **)&d_byteCompressedData, compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_5: %s\n", hipGetErrorString(error));
				
			error = hipMalloc((void **)&d_byteCompressedDataOverflow, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_6: %s\n", hipGetErrorString(error));
							
			// memcpy
			// copy input data
			error = hipMemcpy(d_inputFileData, inputFileData, inputFileLength * sizeof(unsigned char), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_7: %s\n", hipGetErrorString(error));
				
			// copy offset
			error = hipMemcpy(d_compressedDataOffset, compressedDataOffset, (inputFileLength + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_8: %s\n", hipGetErrorString(error));
				
			// copy structure
			error = hipMemcpy(d_huffmanDictionary, &huffmanDictionary, sizeof(huffmanDict), hipMemcpyHostToDevice);
			if (error!= hipSuccess)
					printf("erro_9: %s\n", hipGetErrorString(error));
			
			// initialize d_byteCompressedData
			// initialize bitSequence to byte array to  0
			error = hipMemset(d_byteCompressedData, 0, compressedDataOffset[integerOverflowIndex] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_10: %s\n", hipGetErrorString(error));	
				
			error = hipMemset(d_byteCompressedDataOverflow, 0, compressedDataOffset[inputFileLength] * sizeof(unsigned char));
			if (error!= hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));		
				
			// copy constant memory
			if(constMemoryFlag == 1){
				error = hipMemcpyToSymbol(HIP_SYMBOL(d_bitSequenceConstMemory), bitSequenceConstMemory, 256 * 255 * sizeof(unsigned char));
				if (error!= hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));
			}
		
			// run kernel and copy output
			error = hipMemGetInfo(&mem_free, &mem_total);
			printf("Total GPU space left: %lu\n", mem_free/(1024 * 1024));
			
			// launch kernel
			compress<<<1, block_size>>>(d_inputFileData, d_compressedDataOffset, d_huffmanDictionary, d_byteCompressedData, d_byteCompressedDataOverflow, inputFileLength, constMemoryFlag, integerOverflowIndex);
			
			// check status
			hipError_t error_kernel = hipGetLastError();
			if (error_kernel != hipSuccess)
				printf("erro_final: %s\n", hipGetErrorString(error_kernel));
			
			// get output data
			if(bitPaddingFlag == 0){
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], &d_inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));
			}
			else{
				error = hipMemcpy(inputFileData, d_inputFileData, (compressedDataOffset[integerOverflowIndex] / 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_11: %s\n", hipGetErrorString(error));
				unsigned char temp_huffmanTreeNode = inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1];
				
				error = hipMemcpy(&inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1], &d_inputFileData[(compressedDataOffset[integerOverflowIndex] / 8)], ((compressedDataOffset[inputFileLength] / 8)) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				if (error != hipSuccess)
					printf("erro_12: %s\n", hipGetErrorString(error));				
				inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1] = temp_huffmanTreeNode | inputFileData[(compressedDataOffset[integerOverflowIndex] / 8) - 1];
			}

			hipFree(d_inputFileData);
			hipFree(d_compressedDataOffset);
			hipFree(d_huffmanDictionary);
			hipFree(d_byteCompressedData);
			hipFree(d_byteCompressedDataOverflow);
			
			// write src inputFileLength, header and compressed data to output file
			compressedFile = fopen(*file, "wb");
			fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
			fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
			fwrite(inputFileData, sizeof(unsigned char), (compressedDataOffset[inputFileLength] / 8 + compressedDataOffset[integerOverflowIndex] / 8) - 1, compressedFile);
			fclose(compressedFile);			
		}
	}
	return 0;
}